#include "hip/hip_runtime.h"
#include <stdio.h>
#include "face/cu_model.h"

#define BLOCKSIZE 128

void loadModelToCUDADevice(C_PcaDeformModel *deformModel,
                           const Eigen::MatrixXf deformBasis, const Eigen::VectorXf ref,
                           const std::vector<int> lmkInds) {

    hipMalloc((void**)(&deformModel->deformBasis_d), deformBasis.size()*sizeof(float));
    hipMalloc((void**)(&deformModel->ref_d), ref.size()*sizeof(float));
    hipMalloc((void**)(&deformModel->lmks_d), lmkInds.size()*sizeof(int));

    hipMemcpy((void*)deformModel->deformBasis_d,
               deformBasis.data(), deformBasis.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)deformModel->ref_d,
               ref.data(), ref.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)deformModel->lmks_d,
               lmkInds.data(), lmkInds.size()*sizeof(int), hipMemcpyHostToDevice);
    deformModel->rank = (int)deformBasis.cols();
    deformModel->dim = (int)deformBasis.rows();
    deformModel->lmkCount = (int)lmkInds.size();

    assert(deformBasis.rows() == ref.size());
}

void freeModelCUDA(C_PcaDeformModel deformModel) {
    hipFree(deformModel.deformBasis_d);
    hipFree(deformModel.ref_d);
    hipFree(deformModel.lmks_d);
}

void loadScanToCUDADevice(C_ScanPointCloud *scanPointCloud,
                          boost::shared_ptr<const pcl::PointCloud<pcl::PointXYZRGBA>> scan) {

    hipMalloc((void**)(&scanPointCloud->scanPoints_d), scan->points.size()*3*sizeof(float));

    float *scanPoints = new float[scan->points.size()*3];
    for (int i=0; i<scan->points.size(); i+=3) {
        scanPoints[i] = scan->points[i].x;
        scanPoints[i+1] = scan->points[i].y;
        scanPoints[i+2] = scan->points[i].z;
    }

    hipMemcpy((void*)scanPointCloud->scanPoints_d,
               scanPoints, scan->points.size()*3*sizeof(float), hipMemcpyHostToDevice);
}

void freeScanCUDA(C_ScanPointCloud scanPointCloud) {
    hipFree(scanPointCloud.scanPoints_d);
}

void allocParamsToCUDADevice(C_Params *params, int numParams) {
    hipMalloc((void **)(&params->params_d), numParams*sizeof(float));
    float *zero = new float[numParams]{0,};
    params->numParams = numParams;

    updateParamsInCUDADevice(*params, zero, numParams);
    delete[] zero;
}

void updateParamsInCUDADevice(const C_Params params, const float * const paramsIn, int numParams) {
    hipMemcpy((void*)params.params_d, paramsIn, numParams*sizeof(float), hipMemcpyHostToDevice);
}

void freeParamsCUDA(C_Params params) {
    hipFree(params.params_d);
}

void allocPositionCUDA(float **position_d, int dim) {
    hipMalloc((void**)(position_d), dim*sizeof(float));
}

void freePositionCUDA(float *position_d) {
    hipFree(position_d);
}

__global__
void _calculateVertexPosition(float *position_d, const C_Params params, const C_PcaDeformModel deformModel) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    const int colDim = deformModel.dim;

    position_d[i] = 0;
    for (int j=0; j<deformModel.rank; j++) {
        position_d[i] += params.params_d[j] * deformModel.deformBasis_d[i + colDim * j];
    }
}

void calculateVertexPosition(float *position_d, const C_Params params, const C_PcaDeformModel deformModel) {
    int idim = deformModel.dim;
    dim3 dimBlock(BLOCKSIZE);
    dim3 dimGrid((idim + BLOCKSIZE-1)/BLOCKSIZE);
    _calculateVertexPosition<<<dimGrid, dimBlock>>>(position_d, params, deformModel);
}

void calculateLoss(float *residual, float *jacobian,
                   const float *position_d, const C_Params params,
                   const C_PcaDeformModel deformModel, const C_ScanPointCloud scanPointCloud) {

}
